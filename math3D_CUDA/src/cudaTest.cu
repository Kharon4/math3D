#include "hip/hip_runtime.h"

#include "cudaTest.cuh"
#include "linearMath.cuh"
//#include "vec3.cuh"
//#include "vec3.cu"
#include <stdio.h>

hipError_t addWithCuda(vec3d* c,vec3d *a, unsigned int size);

__global__ void addKernel(vec3d* c,vec3d *a)
{
    int i = threadIdx.x;
    linearMath::line<double> l;
    l.setPT(vec3d(5, 5, 5));
    c[i] = vec3d::normalizeRaw(a[i]);
}

int Main()
{
    const int arraySize = 3;
    const vec3d a[arraySize];
    vec3d b[arraySize] = {vec3d(1,1,1),vec3d(2,2,2), vec3d(3,3,3)};
    vec3d c[arraySize];
    
    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, b , arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    for (int i = 0; i < arraySize; ++i) {
        printf("%f , %f , %f \n", c[i].x, c[i].y, c[i].z);
    }

    
    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(vec3d* c,vec3d * a, unsigned int size)
{
    vec3d* dev = 0;
    vec3d* dev_a = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev, size * sizeof(vec3d));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }
    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(vec3d));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev, c, size * sizeof(vec3d), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(vec3d), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel << <1, size >> > (dev,dev_a);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev, size * sizeof(vec3d), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(a, dev_a, size * sizeof(vec3d), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev);
    return cudaStatus;
}
